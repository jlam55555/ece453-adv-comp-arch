#include "main.h"

// allocate a memory region on the device and copy a value from the host
__host__ void alloc_copy_htd(void *hptr, void **dptr, unsigned size,
	std::string name)
{
	CUDAERR(hipMalloc(dptr, size), "allocating " << name <<);
	CUDAERR(hipMemcpy(*dptr, hptr, size, hipMemcpyHostToDevice),
		"copying " << name << " to device");
}

// copy memory region from device to host (if hptr specified)
__host__ void copy_dth(void *hptr, void *dptr, unsigned size,
	std::string name)
{
	CUDAERR(hipMemcpy(hptr, dptr, size, hipMemcpyDeviceToHost),
		"copying " << name <<);
}

// free emory region on device
__host__ void free_d(void *dptr, std::string name)
{
	CUDAERR(hipFree(dptr), "freeing " << name <<);
}
