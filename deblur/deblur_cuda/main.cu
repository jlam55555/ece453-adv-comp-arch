#include "hip/hip_runtime.h"
#include <cmath>
#include "main.h"
#include "pngio.h"

// these are declared in main.h
hipError_t err = hipSuccess;
float *dImg, *dTmp1, *dTmp2, *dTmp3;
unsigned int rowStride, channels, bufSize, blockSize;
dim3 dimGrid, dimBlock;

// copy d1 to d2, but change from unsigned char to float
__global__ static void byteToFloat(byte *d1, float *d2, int h, int rs)
{
	unsigned int y, x;

	// infer y, x, c from block/thread index
	y = blockDim.y * blockIdx.y + threadIdx.y;
	x = blockDim.x * blockIdx.x + threadIdx.x;
	if (y >= h || x >= rs) {
		return;
	}

	d2[y*rs + x] = d1[y*rs + x];
}

// copy d1 to d2, but change from float to unsigned char
__global__ static void floatToByte(float *d1, byte *d2, int h, int rs)
{
	unsigned int y, x;

	// infer y, x, c from block/thread index
	y = blockDim.y * blockIdx.y + threadIdx.y;
	x = blockDim.x * blockIdx.x + threadIdx.x;
	if (y >= h || x >= rs || (x%4==3)) {
		return;
	}

	d2[y*rs + x] = min(max(d1[y*rs + x], 0.), 255.);
}

// image processing routines go here
__host__ static void processImage(void)
{
	// perform 25 iterations of the LR deconvolution
	// with a gaussian kernel with sigma=2
	deblur(25, 2);
}

// driver for function
__host__ int main(int argc, char **argv)
{
	// allocate buffers for image, copy into contiguous array
	byte *hImgPix = nullptr, *dImgPix = nullptr;
	clock_t *t;
	unsigned int y;

	// get input file from stdin
	ERR(argc < 3, "usage: ./deblur INPUT.png OUTPUT.png");

	// read input file
	std::cout << "Reading file..." << std::endl;
	read_png_file(argv[1]);

	// assume only RGB (3 channels) or RGBA (4 channels)
	channels = color_type==PNG_COLOR_TYPE_RGBA ? 4 : 3;
	rowStride = width * channels;
	bufSize = rowStride * height;

	// allocate host buffer, copy image to buffers
	ERR(!(hImgPix = (byte *) malloc(bufSize)),
		"allocating contiguous buffer for image");

	// allocate other buffers
	CUDAERR(hipMalloc((void **) &dImgPix, bufSize), "allocating dImgPix");
	CUDAERR(hipMalloc((void **) &dImg, bufSize*sizeof(float)),
		"allocating dImg");
	CUDAERR(hipMalloc((void **) &dTmp1, bufSize*sizeof(float)),
		"allocating dTmp1");
	CUDAERR(hipMalloc((void **) &dTmp2, bufSize*sizeof(float)),
		"allocating dTmp2");
	CUDAERR(hipMalloc((void **) &dTmp3, bufSize*sizeof(float)),
		"allocating dTmp3");

	// copy image to contiguous buffer (double pointer is not guaranteed
	// to be contiguous)
	for (y = 0; y < height; ++y) {
		memcpy(hImgPix+rowStride*y, row_pointers[y], rowStride);
	}

	// copy image to device (hImgPix -> dImgPix)
	CUDAERR(hipMemcpy(dImgPix, hImgPix, bufSize, hipMemcpyHostToDevice),
		"copying image to device");

	// set kernel parameters (same for all future kernel invocations)
	blockSize = 32;
	dimGrid = dim3(ceil(rowStride*1./blockSize),
		ceil(height*1./blockSize), 1);
	dimBlock = dim3(blockSize, blockSize, 1);

	// convert image to float (dImgPix -> dImg)
	byteToFloat<<<dimGrid, dimBlock>>>(dImgPix, dImg, height, rowStride);
	CUDAERR(hipGetLastError(), "launch byteToFloat kernel");

	// image processing routine
	std::cout << "Processing image..." << std::endl;
	t = clock_start();
	processImage();
	clock_lap(t, CLOCK_OVERALL);

	// print statistics
	std::cout << "overall: " << clock_ave[CLOCK_OVERALL] << "s" << std::endl
		<< "round: " << clock_ave[CLOCK_ROUND] << "s" << std::endl
		<< "conv2d: " << clock_ave[CLOCK_CONV2D] << "s" << std::endl
		<< "mult/div: " << clock_ave[CLOCK_MULTDIV] << "s" << std::endl;

	// convert image back to byte (dImg -> dImgPix)
	floatToByte<<<dimGrid, dimBlock>>>(dImg, dImgPix, height, rowStride);
	CUDAERR(hipGetLastError(), "launch floatToByte kernel");

	// copy image back (dImgPix -> hImgPix)
	CUDAERR(hipMemcpy(hImgPix, dImgPix, bufSize, hipMemcpyDeviceToHost),
		"copying image from device");

	// copy image back into original pixel buffers
	for (y = 0; y < height; ++y) {
		memcpy(row_pointers[y], hImgPix+rowStride*y, rowStride);
	}

	// free buffers
	CUDAERR(hipFree(dImg), "freeing dImg");
	CUDAERR(hipFree(dTmp1), "freeing dTmp1");
	CUDAERR(hipFree(dTmp2), "freeing dTmp2");
	CUDAERR(hipFree(dTmp3), "freeing dTmp2");
	CUDAERR(hipFree(dImgPix), "freeing dImgPix");
	free(hImgPix);
	free(t);

	// write file
	std::cout << "Writing file..." << std::endl;
	write_png_file(argv[2]);

	std::cout << "Done." << std::endl;
	return 0;
}
