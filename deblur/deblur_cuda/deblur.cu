#include "hip/hip_runtime.h"
#include "main.h"

// pointwise multiplication/division of two vectors
__global__ static void pointwiseMultDiv(float *dA, float *dB, float *dC,
	int height, int rowStride, int channels, bool isMult)
{
	unsigned y, x, ind;

	// infer y, x, c from block/thread index
	y = blockDim.y * blockIdx.y + threadIdx.y;
	x = blockDim.x * blockIdx.x + threadIdx.x;
	if (y >= height || x >= rowStride) {
		return;
	}

	ind = y*rowStride + x;

	// full alpha
	if (channels==4 && x%4==3) {
		dC[ind] = 255;
		return;
	}

	dC[ind] = isMult
		? dA[ind] * dB[ind] / 255.
		: dA[ind] / max(dB[ind],1.) * 255.;	// prevent /0
}

// set image to median color
__global__ static void initImage(float *dImg, int height, int rowStride,
	int channels)
{
	unsigned y, x;

	// infer y, x, c from block/thread index
	y = blockDim.y * blockIdx.y + threadIdx.y;
	x = blockDim.x * blockIdx.x + threadIdx.x;
	if (y >= height || x >= rowStride) {
		return;
	}

	dImg[y*rowStride + x] = (channels==4 && x%4==3) ? 255 : 127;
}

// perform one "round" of LR deconvolution
__host__ static void deblurRound(float *g, unsigned fltSize)
{
	float *tmp;
	unsigned i, j;

	// convolution: tmp3 = f_i * g
	// dTmp3 = dTmp1 * flt
	conv2d<<<dimGrid, dimBlock>>>(dTmp1, g, dTmp3, channels,
		height, width, fltSize, fltSize);
	CUDAERR(hipGetLastError(), "launch conv2d kernel 1");

	// pointwise division: tmp2 = c / tmp3
	// dTmp2 = dImg / dTmp3
	pointwiseMultDiv<<<dimGrid, dimBlock>>>(dImg, dTmp3, dTmp2, height,
		rowStride, channels, false);
	CUDAERR(hipGetLastError(), "launch div kernel");

	// convolution: tmp3 = tmp2 * g(-x) = tmp2 * g (g is symmetric)
	// dTmp3 = dTmp2 * g
	conv2d<<<dimGrid, dimBlock>>>(dTmp2, g, dTmp3, channels,
		height, width, fltSize, fltSize);
	CUDAERR(hipGetLastError(), "launch conv2d kernel 2");

	// pointwise multiplication: tmp2 = (tmp3)(f_i)
	// dTmp2 = (dTmp3)(dTmp1)
	pointwiseMultDiv<<<dimGrid, dimBlock>>>(dTmp3, dTmp1, dTmp2, height,
		rowStride, channels, true);
	CUDAERR(hipGetLastError(), "launch mult kernel");

	// swap pointers so that f_i = dTmp1
	// dTmp2, dTmp1 = dTmp1, dTmp2
	tmp = dTmp2;
	dTmp2 = dTmp1;
	dTmp1 = tmp;
}

// lucy richardson deblur: deblurs what is in dImg
__host__ void deblur(int rounds, int blurSize)
{
	float *hFlt, *dFlt, *tmp;
	unsigned fltSize, i;

	// initialize f_0 (initial estimate)
	initImage<<<dimGrid, dimBlock>>>(dTmp1, height, rowStride, channels);
	CUDAERR(hipGetLastError(), "launch initImage kernel");

	// get initial gaussian filter
	gaussian_filter(blurSize, &hFlt, &fltSize);

	// allocate and copy filter to device
	alloc_copy_htd(hFlt, (void **) &dFlt, fltSize*fltSize*sizeof(float),
		"flt");

	// lucy-richardson iteration
	for (i = 0; i < rounds; ++i) {
		deblurRound(dFlt, fltSize);
	}

	// dTmp1 is currently pointing at f_i (the estimate)
	tmp = dTmp1;
	dTmp1 = dImg;
	dImg = tmp;

	// cleanup
	free(hFlt);
	free_d(dFlt, "dFlt");
}
